
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 5
__global__ void add(int *a, int *b, int *c) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < N) {
c[i] = a[i] + b[i];
}
}
int main() {
int a[N] = {1, 2, 3, 4, 5};
int b[N] = {6, 7, 8, 9, 10};
int c[N] = {0};
int *dev_a, *dev_b, *dev_c;
hipMalloc((void **)&dev_a, N * sizeof(int));
hipMalloc((void **)&dev_b, N * sizeof(int));
hipMalloc((void **)&dev_c, N * sizeof(int));
hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
add<<<1, N>>>(dev_a, dev_b, dev_c);
hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
for (int i = 0; i < N; i++) {
//printf("%d ", c[i]);
printf("%d + %d = %d\n", a[i], b[i], c[i]);
}
printf("\n");
hipFree(dev_a);
hipFree(dev_b);
hipFree(dev_c);
return 0;
}